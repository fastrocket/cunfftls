#include "hip/hip_runtime.h"
/*   cunfftls_utils.cu
 *   =================
 *   
 *   Misc. functions useful for other parts of the program 
 * 
 *   (c) 2016, John Hoffman
 *   code borrowed extensively from B. Leroy's nfftls
 * 
 *   This file is part of cunfftls
 *
 *   cunfftls is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   cunfftls is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with cunfftls.  If not, see <http://www.gnu.org/licenses/>.
 *  
 *   Copyright (C) 2016, John Hoffman, 2012 B. Leroy [nfftls]
 */

#include <stdlib.h>
#include "cuna_utils.h"
#include "cuna_filter.h"
#include "cunfftls_utils.h"
#include "cunfftls_typedefs.h"


#ifdef DOUBLE_PRECISION
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
			old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

__host__ void getNfreqsAndCorrOversampling(int npts, Settings *settings){
   dTyp nfreqsr = 0.5 * npts * settings->over * settings->hifac;

   // correct the "oversampling" parameter accordingly
   settings->nfreqs = (int) nextPowerOfTwo(floor(nfreqsr));
   settings->over  *= settings->nfreqs / nfreqsr;

   //fprintf(stderr, "nfreqsr = %.5e, poweof2 = %.5e, nfreqs = %d\n", nfreqsr, nextPowerOfTwo(nfreqsr), settings->nfreqs);

}

__global__ void
convertToComplex(dTyp *a, Complex *c, int N){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		c[i].x = a[i];
		c[i].y = 0;
    }
}

__host__ dTyp 
nextPowerOfTwo(dTyp v) {
	return round(pow(2, round(log10(v) / log10(2.0)) + 1));
}

__host__ void
set_device(int device) {
	checkCudaErrors(hipSetDevice(device));
}

__host__ void
meanAndVariance(int n, const dTyp *y, dTyp *mean , dTyp *variance) {
  *mean = 0;
  dTyp M2 = 0, delta;
  
  int nn = 1;
  for(int i = 0; i < n; i++, nn++) {
    delta = y[i] - *mean;
    *mean += delta / nn;
    M2 += delta * (y[i] - *mean);
  }
  *variance = M2/(n - 1);
}
__device__ dTyp
sign(dTyp a, dTyp b) {
  	return ((b >= 0) ? 1 : -1) * absoluteValueReal(a);
}

__device__ dTyp
square(dTyp a) { 
	return a * a; 
}


// converts clock_t value into seconds
__host__ dTyp 
seconds(clock_t dt) {
	return ((dTyp) dt) / ((dTyp)CLOCKS_PER_SEC);
}

