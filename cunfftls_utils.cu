#include "hip/hip_runtime.h"
/*   cunfftls_utils.cu
 *   =================
 *   
 *   Misc. functions useful for other parts of the program 
 * 
 *   (c) 2016, John Hoffman
 *   code borrowed extensively from B. Leroy's nfftls
 * 
 *   This file is part of cunfftls
 *
 *   cunfftls is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   cunfftls is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with cunfftls.  If not, see <http://www.gnu.org/licenses/>.
 *  
 *   Copyright (C) 2016, John Hoffman, 2012 B. Leroy [nfftls]
 */

#include <stdlib.h>
#include "cuna_utils.h"
#include "cuna_filter.h"
#include "cunfftls_utils.h"
#include "cunfftls_typedefs.h"


__host__ void getNfreqsAndCorrOversampling(int npts, Settings *settings){
   dTyp nfreqsr = 0.5 * npts * settings->over0 * settings->hifac;

   // correct the "oversampling" parameter accordingly
   settings->nfreqs = (int) nextPowerOfTwo(floor(nfreqsr));
   settings->over   = settings->over0 * settings->nfreqs / nfreqsr;

}

__global__ void
convertToComplex(dTyp *a, Complex *c, int N){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		c[i].x = a[i];
		c[i].y = 0;
    }
}

__host__ dTyp 
nextPowerOfTwo(dTyp v) {
	return round(pow(2, round(log10(v) / log10(2.0)) + 1));
}

__host__ void
set_device(int device) {
	checkCudaErrors(hipSetDevice(device));
}

__host__ void
meanAndVariance(const int n, const dTyp *y, dTyp *mean , dTyp *variance) {
  *mean = 0;
  dTyp M2 = 0, delta;
  
  int nn = 1;
  for(int i = 0; i < n; i++, nn++) {
    delta = y[i] - *mean;
    *mean += delta / nn;
    M2 += delta * (y[i] - *mean);
  }
  *variance = M2/(n - 1);
}

__host__ void
weightedMeanAndVariance(const int n, const dTyp *y, const dTyp *w, 
                        dTyp *mean, dTyp *variance) {
  *mean = 0;
  *variance = 0;

  for(int i = 0; i < n; i++) 
    *mean += w[i] * y[i];
  
  for(int i = 0; i < n; i++) 
    *variance += w[i] * (y[i] - *mean) * (y[i] - *mean);
  
}

__device__ dTyp
sign(dTyp a, dTyp b) {
  	return ((b >= 0) ? 1 : -1) * absoluteValueReal(a);
}

__device__ dTyp
square(dTyp a) { 
	return a * a; 
}


// converts clock_t value into seconds
__host__ dTyp 
seconds(clock_t dt) {
	return ((dTyp) dt) / ((dTyp)CLOCKS_PER_SEC);
}

__host__ void
randomSample(const int npts, const dTyp *tobs, const dTyp *yobs, 
              const dTyp *erobs, dTyp *t, dTyp *y, dTyp *er) {
  int u;
  if (erobs == NULL) er = NULL;
  
  for (int i = 0; i < npts; i++) {
    u = rand() % npts;
    t[i] = tobs[u];
    y[i] = yobs[u];
    if (erobs != NULL)
      er[i] = erobs[u];
  }
}
